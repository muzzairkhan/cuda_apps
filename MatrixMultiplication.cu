#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define A_KILO_BYTES (1024)
/* The dimensions of matrices, keep it at least 16. */
#define V_SIZE (4 * A_KILO_BYTES)

/* Populate the input matrices with random numbers between 0 and 99. */
void populateInputMatrix(int a[V_SIZE][V_SIZE], int b[V_SIZE][V_SIZE])
{
    for (int i = 0; i < V_SIZE; i++)
    {
        for (int j = 0; j < V_SIZE; j++)
        {
            a[i][j] = rand() % 99;
            b[i][j] = rand() % 99;
        }
    }
}

/* For visual verification. */
void printMatrix(int m[V_SIZE][V_SIZE])
{
    printf("Matrix: \n");
    for (int i = 0; i < V_SIZE; i++)
    {
        for (int j = 0; j < V_SIZE; j++)
        {
            printf("%d ", m[i][j]);
        }
        printf("\n");
    }
}

void multiplyWithCPU(int c[V_SIZE][V_SIZE], const int a[V_SIZE][V_SIZE], const int b[V_SIZE][V_SIZE])
{
    for (int i = 0; i < V_SIZE; i++) // Chose a row of A
    {
        for (int j = 0; j < V_SIZE; j++) // Chose a col of B
        {
            for (int k = 0; k < V_SIZE; k++) // loop over the elements of chosen row and chosen col
            {
                c[i][j] += a[i][k] * b[k][j];
            }
        }

    }
}

bool validateResults(const int a[V_SIZE][V_SIZE], const int b[V_SIZE][V_SIZE])
{
    {
        for (int i = 0; i < V_SIZE; i++)
        {
            for (int j = 0; j < V_SIZE; j++)
            {
                if (a[i][j] != b[i][j])
                    return false;
            }
        }
    }
    return true;
}

__global__ void multiplyMatricesKernel(int* c, const int* a, const int* b, size_t pitch)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; // a row number, we shall use it for indexing matrix a
    int j = blockIdx.x * blockDim.x + threadIdx.x; //  a column number, we shall use it for indexing matrix b
    int val = 0;

    if (i < V_SIZE && j < V_SIZE) // for the threads which don't have data to process
    {
        for (int k = 0; k < V_SIZE; k++)
        {
            val += a[(i * V_SIZE) + k] * b[j + (k * V_SIZE)];
        }
        c[j + (i * V_SIZE)] = val;
    }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t multiplyWithCuda(int c[V_SIZE][V_SIZE], const int a[V_SIZE][V_SIZE], const int b[V_SIZE][V_SIZE])
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    size_t pitch;
    hipError_t cudaStatus;
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(V_SIZE / threadsPerBlock.x, V_SIZE / threadsPerBlock.y);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)
    cudaStatus = hipMallocPitch(&dev_a, &pitch, V_SIZE * sizeof(int), V_SIZE);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_a failed!");
        goto Error;
    }

    cudaStatus = hipMallocPitch(&dev_b, &pitch, V_SIZE * sizeof(int), V_SIZE);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_b failed!");
        goto Error;
    }

    cudaStatus = hipMallocPitch(&dev_c, &pitch, V_SIZE * sizeof(int), V_SIZE);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_c failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy2D(dev_a, pitch, a, V_SIZE * sizeof(int), V_SIZE * sizeof(int), V_SIZE, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy dev_a failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy2D(dev_b, pitch, b, V_SIZE * sizeof(int), V_SIZE * sizeof(int), V_SIZE, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy dev_b failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    multiplyMatricesKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b, pitch);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyMatricesKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy2D(c, V_SIZE * sizeof(int), dev_c, pitch, V_SIZE * sizeof(int), V_SIZE, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy dev_c failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

int main()
{
    int a[V_SIZE][V_SIZE] = { 0 };
    int b[V_SIZE][V_SIZE] = { 0 };
    int c[V_SIZE][V_SIZE] = { 0 };
    int resultWithCPU[V_SIZE][V_SIZE] = { 0 };

    // For random values generation to populate input vectors.
    srand((unsigned int)time(NULL));

    populateInputMatrix(a, b);
    //printMatrix(a);
    //printMatrix(b);

    // Multiply Matrices at host CPU.
    multiplyWithCPU(resultWithCPU, a, b);
    //printMatrix(resultWithCPU);

    // Multiply vectors in parallel.
    hipError_t cudaStatus = multiplyWithCuda(c, a, b);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyWithCuda failed!");
        return 1;
    }

    // Match the the vector addition results from both CPU and GPU.
    if (!validateResults(c, resultWithCPU))
    {
        fprintf(stderr, "<<<< ERROR: Matrix multiplication failed with CUDA. >>>> \n");
        return 1;
    }
    else
        printf("MATRIX MUTIPLICATION SUCCESS.\n");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}